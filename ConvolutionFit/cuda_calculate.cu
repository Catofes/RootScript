#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/transform.h>
#include <thrust/sequence.h>
#include <thrust/copy.h>
#include <thrust/fill.h>
#include <thrust/replace.h>
#include <thrust/functional.h>
#include <iostream>
#include <thrust/complex.h>
#include "cuda_calculate.h"
#include <chrono>
#include <hip/hip_runtime.h>

#define GPUDEBUG

struct sub_f
{
    const double max;
    const double min;
    const double mean;
    const double width;


    sub_f(double _max, double _min, double _mean, double _width) : max(_max), min(_min), mean(_mean), width(_width)
    {}

    __device__
    double operator()(const double &x) const
    {
        double _x = x;
        if (x > max) {
            _x = 2 * max - x;
        }
        if (x < min) {
            _x = 2 * min - x;
        }
        double w = (width > 0) ? width : -width;
        return 1. / ((_x - mean) * (_x - mean) + 0.25 * w * w);
    }
};

struct sub_sigma
{
    const double sigma;

    sub_sigma(double _sigma) : sigma(_sigma)
    {}

    __device__
    double operator()(const double &x) const
    {
        return sigma;
    }
};

struct sub_gauss
{
    const double t;

    sub_gauss(double _t) : t(_t)
    {}

    __device__
    double operator()(const double &x, const double &sigma) const
    {
//        thrust::complex<double> v(-0.5 / (sigma * sigma) * (x - t) * (x - t), 0);
//        return thrust::exp(v).real() / sigma;
        return exp(-0.5 / (sigma * sigma) * (x - t) * (x - t));
    }
};


thrust::device_vector<double> *d_t = nullptr;
thrust::device_vector<double> *d_sigma = nullptr;
thrust::device_vector<double> *d_x = nullptr;
thrust::device_vector<double> *d_w = nullptr;

double sub_cuda_normal_calculate(int bins, double min, double max, double x, double mean, double width, double f_min,
                                 double f_max)
{
#ifdef GPUDEBUG
    std::chrono::system_clock::time_point start, finish;
#endif

    if (d_t == nullptr) {
        d_t = new thrust::device_vector<double>(bins);
    }
    if (d_sigma == nullptr) {
        d_sigma = new thrust::device_vector<double>(bins);
    }
#ifdef GPUDEBUG
    start = std::chrono::high_resolution_clock::now();
#endif

    thrust::sequence((*d_t).begin(), (*d_t).end(), min, (max - min) / bins);

#ifdef GPUDEBUG
    finish = std::chrono::high_resolution_clock::now();
    std::cout << "s1: " << std::chrono::duration_cast<std::chrono::nanoseconds>(finish - start).count() << std::endl;
    start = std::chrono::high_resolution_clock::now();
#endif

    thrust::transform((*d_t).begin(), (*d_t).end(), (*d_sigma).begin(), sub_sigma(0.5));

#ifdef GPUDEBUG
    finish = std::chrono::high_resolution_clock::now();
    std::cout << "s2: " << std::chrono::duration_cast<std::chrono::nanoseconds>(finish - start).count() << std::endl;
    start = std::chrono::high_resolution_clock::now();
#endif

    thrust::transform((*d_t).begin(), (*d_t).end(), (*d_sigma).begin(), (*d_sigma).begin(), sub_gauss(x));

#ifdef GPUDEBUG
    finish = std::chrono::high_resolution_clock::now();
    std::cout << "s3: " << std::chrono::duration_cast<std::chrono::nanoseconds>(finish - start).count() << std::endl;
    start = std::chrono::high_resolution_clock::now();
#endif

    thrust::transform((*d_t).begin(), (*d_t).end(), (*d_t).begin(), sub_f(f_max, f_min, mean, width));

#ifdef GPUDEBUG
    finish = std::chrono::high_resolution_clock::now();
    std::cout << "s4: " << std::chrono::duration_cast<std::chrono::nanoseconds>(finish - start).count() << std::endl;
    start = std::chrono::high_resolution_clock::now();
#endif

    thrust::transform((*d_t).begin(), (*d_t).end(), (*d_sigma).begin(), (*d_t).begin(), thrust::multiplies<double>());

#ifdef GPUDEBUG
    finish = std::chrono::high_resolution_clock::now();
    std::cout << "s5: " << std::chrono::duration_cast<std::chrono::nanoseconds>(finish - start).count() << std::endl;
    start = std::chrono::high_resolution_clock::now();
#endif

    double result = thrust::reduce((*d_t).begin(), (*d_t).end(), double(0), thrust::plus<double>());

#ifdef GPUDEBUG
    finish = std::chrono::high_resolution_clock::now();
    std::cout << "s6: " << std::chrono::duration_cast<std::chrono::nanoseconds>(finish - start).count() << std::endl;
    start = std::chrono::high_resolution_clock::now();
#endif

    return result;
}

void sub_cuda_gaus_prepare(std::vector<double> &x, std::vector<double> &w, int bins)
{
    if (d_x != nullptr) {
        delete d_x;
    }
    if (d_w != nullptr) {
        delete d_w;
    }
    d_x = new thrust::device_vector<double>(bins);
    d_w = new thrust::device_vector<double>(bins);
    thrust::copy(x.begin(), x.end(), (*d_x).begin());
    thrust::copy(w.begin(), w.end(), (*d_w).begin());
}

double sub_cuda_gaus_calculate(int bins, double min, double max, double x, double mean, double width, double f_min,
                               double f_max)
{
#ifdef GPUDEBUG
    std::chrono::system_clock::time_point start, finish;
#endif

    if (d_t == nullptr) {
        d_t = new thrust::device_vector<double>(bins);
    }
    if (d_sigma == nullptr) {
        d_sigma = new thrust::device_vector<double>(bins);
    }
#ifdef GPUDEBUG
    start = std::chrono::high_resolution_clock::now();
#endif

    thrust::copy((*d_x).begin(), (*d_x).end(), (*d_t).begin());

#ifdef GPUDEBUG
    finish = std::chrono::high_resolution_clock::now();
    std::cout << "s1: " << std::chrono::duration_cast<std::chrono::nanoseconds>(finish - start).count() << std::endl;
    start = std::chrono::high_resolution_clock::now();
#endif

    thrust::transform((*d_t).begin(), (*d_t).end(), (*d_sigma).begin(), sub_sigma(0.5));

#ifdef GPUDEBUG
    finish = std::chrono::high_resolution_clock::now();
    std::cout << "s2: " << std::chrono::duration_cast<std::chrono::nanoseconds>(finish - start).count() << std::endl;
    start = std::chrono::high_resolution_clock::now();
#endif

    thrust::transform((*d_t).begin(), (*d_t).end(), (*d_sigma).begin(), (*d_sigma).begin(), sub_gauss(x));

#ifdef GPUDEBUG
    finish = std::chrono::high_resolution_clock::now();
    std::cout << "s3: " << std::chrono::duration_cast<std::chrono::nanoseconds>(finish - start).count() << std::endl;
    start = std::chrono::high_resolution_clock::now();
#endif

    thrust::transform((*d_t).begin(), (*d_t).end(), (*d_t).begin(), sub_f(f_max, f_min, mean, width));

#ifdef GPUDEBUG
    finish = std::chrono::high_resolution_clock::now();
    std::cout << "s4: " << std::chrono::duration_cast<std::chrono::nanoseconds>(finish - start).count() << std::endl;
    start = std::chrono::high_resolution_clock::now();
#endif

    thrust::transform((*d_t).begin(), (*d_t).end(), (*d_sigma).begin(), (*d_t).begin(), thrust::multiplies<double>());
    thrust::transform((*d_t).begin(), (*d_t).end(), (*d_w).begin(), (*d_t).begin(), thrust::multiplies<double>());

#ifdef GPUDEBUG
    finish = std::chrono::high_resolution_clock::now();
    std::cout << "s5: " << std::chrono::duration_cast<std::chrono::nanoseconds>(finish - start).count() << std::endl;
    start = std::chrono::high_resolution_clock::now();
#endif

    double result = thrust::reduce((*d_t).begin(), (*d_t).end(), double(0), thrust::plus<double>());

#ifdef GPUDEBUG
    finish = std::chrono::high_resolution_clock::now();
    std::cout << "s6: " << std::chrono::duration_cast<std::chrono::nanoseconds>(finish - start).count() << std::endl;
    start = std::chrono::high_resolution_clock::now();
#endif
    return result;
}